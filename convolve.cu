#include "hip/hip_runtime.h"
#include <matrix.h>
#include <mex.h>
#include "gpu/mxGPUArray.h"

/* Definitions to keep compatibility with earlier versions of ML */
#ifndef MWSIZE_MAX
typedef int mwSize;
typedef int mwIndex;
typedef int mwSignedIndex;

#if (defined(_LP64) || defined(_WIN64)) && !defined(MX_COMPAT_32)
/* Currently 2^48 based on hardware limitations */
# define MWSIZE_MAX    281474976710655UL
# define MWINDEX_MAX   281474976710655UL
# define MWSINDEX_MAX  281474976710655L
# define MWSINDEX_MIN -281474976710655L
#else
# define MWSIZE_MAX    2147483647UL
# define MWINDEX_MAX   2147483647UL
# define MWSINDEX_MAX  2147483647L
# define MWSINDEX_MIN -2147483647L
#endif
#define MWSIZE_MIN    0UL
#define MWINDEX_MIN   0UL
#endif


/*
 * Device code
 */
void __global__ TimesTwo(double const * const A,
                         double * const B,
                         int const N)
{
    /* Calculate the global linear index, assuming a 1-d grid. */
    int const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        B[i] = 2.0 * A[i];
    }
}

void __global__ convolution(double const * const A,double const * const B,double * const C,int const M,int const N,int const K)
{
	int const col = blockDim.x * blockIdx.x + threadIdx.x;
	int const row = blockDim.y * blockIdx.y + threadIdx.y;

	if((row<N)&&(col<M))
	{
		double sum = 0;
		for(int i=0;i<K;i++)
		{
			for(int j=0;j<K;j++)
			{
				int x = row+i+(1-K)/2;
				int y = col+j+(1-K)/2;
				sum = sum + B[j*K+i]*((x<N&&x>=0&&y<M&&y>=0)?A[y*N+x]:0);
			}
		}
		C[col*N+row] = sum;
	}	
}


/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    mxArray *A, *B;
    double *a, *b;
    mxGPUArray const *a_in_mn;
    mxGPUArray const *b_in_k;
    mxGPUArray *c_out_mn;
    double const *d_a;
    double const *d_b;
    double *d_c;
    const mwSize *dims_mn, *dims_k;
    int m, n, k, i, j;

    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* Associate inputs */
    a_in_mn = mxGPUCreateFromMxArray(prhs[0]);
    b_in_k = mxGPUCreateFromMxArray(prhs[1]);

    A = mxDuplicateArray(prhs[0]);
    B = mxDuplicateArray(prhs[1]);

    /* Figure out dimensions */
    dims_mn = mxGPUGetDimensions(a_in_mn);
    n = (int)dims_mn[0]; m = (int)dims_mn[1];

    dims_k = mxGPUGetDimensions(b_in_k);
    k = (int)dims_k[0];
    
    /*
     * Verify that a_in_mn, b_in_k  really is a double array before extracting the pointer.
     */
    if (mxGPUGetClassID(a_in_mn) != mxDOUBLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }
    if (mxGPUGetClassID(b_in_k) != mxDOUBLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    /*
     * Now that we have verified the data type, extract a pointer to the input
     * data on the device.
     */
    d_a = (double const *)(mxGPUGetDataReadOnly(a_in_mn));
    d_b = (double const *)(mxGPUGetDataReadOnly(b_in_k));

    a = mxGetPr(A);
    b = mxGetPr(B);
    /*
    mexPrintf("Input Matrix:\n");
    for(i=0;i<n;i++)
    {
        for(j=0;j<m;j++)
        {
            mexPrintf("%f ",a[j*n+i]);
        }
        mexPrintf("\n");
    }
    mexPrintf("Input Kernel:\n");
    for(i=0;i<k;i++)
    {
        for(j=0;j<k;j++)
        {
            mexPrintf("%f ",b[j*k+i]);
        }
        mexPrintf("\n");
    }
    */
    /* Create a GPUArray to hold the result and get its underlying pointer. */
    c_out_mn = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(a_in_mn),
                            	   mxGPUGetDimensions(a_in_mn),
                            	   mxGPUGetClassID(a_in_mn),
                            	   mxGPUGetComplexity(a_in_mn),
                            	   MX_GPU_DO_NOT_INITIALIZE);
    d_c = (double *)(mxGPUGetData(c_out_mn));

    /* Associate outputs */
    plhs[0] = mxGPUCreateMxArrayOnGPU(c_out_mn);


    mexPrintf("Convolving now..\n");
    /* Call kernel here */

    dim3 DimGrid((m-1)/16+1,(n-1)/16+1,1);
    dim3 DimBlock(16,16,1);

    mexPrintf("CUDA kernel launch with %d blocks of %d threads\n", DimGrid.x*DimGrid.y*DimGrid.z, DimBlock.x*DimBlock.y*DimBlock.z);

    convolution<<<DimGrid, DimBlock>>>(d_a, d_b, d_c, m, n, k);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(a_in_mn);
    mxGPUDestroyGPUArray(b_in_k);
    mxGPUDestroyGPUArray(c_out_mn);
}
